#include "hip/hip_runtime.h"
#include "MatrixLib/coloring.cuh"
#include "UtilLib/utils.cuh"
#include "UtilLib/cuda_utils.hpp"

__global__ void color_for_forward_pass_kernel(
    int num_rows, int num_stripes, int diag_offset, double * A, int * j_min_i, int * colors
){

    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // we initialize the first row to be colored with 0
    if(tid == 0){
        colors[0] = 0;
    }
    
    // we loop over all the rows
    for(int i = tid; i < num_rows; i += gridDim.x * blockDim.x){
        int iterations_ctr = 0;
        
        int my_color = colors[i];
        // as long as our row is not colored we loop
        while(my_color < 0){

        // printf("from thread %d, block %d, i = %d, my_color %d \n", threadIdx.x, blockIdx.x, i, my_color);
            iterations_ctr++;
            int max_color = -1;
            int min_color = 0;

            // we loop over all the stripes, checking if we can color the row
            for(int stripe = 0; stripe < diag_offset; stripe++){
                int j = j_min_i[stripe] + i;
                double val = A[i * num_stripes + stripe];

                if(j < num_rows && j >= 0 && val != 0.0){
                    int color = colors[j];
                    
                    if(color > max_color){
                        max_color = color;
                    }
                    if(color < min_color){
                        min_color = color;
                    }
                }
            }

            if (min_color >= 0){
                // this means we can color the row
                my_color = max_color + 1;
                atomicExch(&colors[i], my_color);
                // printf("from thread %d, block %d, i = %d, my_color %d \n", threadIdx.x, blockIdx.x, i, my_color);
            }
        }
    }
}

__global__ void color_for_backward_pass_kernel(
    int num_rows, int num_stripes, int diag_offset, double * A, int * j_min_i, int * colors
){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // we initialize the last row to be colored with 0
    if(tid == 0){
        colors[num_rows-1] = 0;
    }
    
    // we loop over all the rows
    for(int i = num_rows - tid-1; i >= 0; i -= gridDim.x * blockDim.x){
        int iterations_ctr = 0;
        
        int my_color = colors[i];
        // as long as our row is not colored we loop
        while(my_color < 0){

        // printf("from thread %d, block %d, i = %d, my_color %d \n", threadIdx.x, blockIdx.x, i, my_color);
            iterations_ctr++;
            int max_color = -1;
            int min_color = 0;

            // we loop over all the stripes, checking if we can color the row
            for(int stripe = diag_offset + 1; stripe < num_stripes; stripe++){
                int j = j_min_i[stripe] + i;
                double val = A[i * num_stripes + stripe];

                if(j < num_rows && j >= 0 && val != 0.0){
                    int color = colors[j];
                    
                    if(color > max_color){
                        max_color = color;
                    }
                    if(color < min_color){
                        min_color = color;
                    }
                }
            }

            if (min_color >= 0){
                // this means we can color the row
                my_color = max_color + 1;
                atomicExch(&colors[i], my_color);
                // printf("from thread %d, block %d, i = %d, my_color %d \n", threadIdx.x, blockIdx.x, i, my_color);
            }
        }
    }
}

std::vector<int> color_for_forward_pass(striped_Matrix <double> A){

    int num_rows = A.get_num_rows();
    int num_stripes = A.get_num_stripes();
    int diag_offset = A.get_diag_index();

    std::vector<int> colors(num_rows, -1);

    // put everything on the device
    int * colors_d;
    int * j_min_i_d;
    double * A_d;

    CHECK_CUDA(hipMalloc(&colors_d, num_rows * sizeof(int)));
    CHECK_CUDA(hipMalloc(&j_min_i_d, num_rows * sizeof(int)));
    CHECK_CUDA(hipMalloc(&A_d, num_stripes * num_rows * sizeof(double)));

    CHECK_CUDA(hipMemset(colors_d, -1, num_rows * sizeof(int)));
    CHECK_CUDA(hipMemcpy(j_min_i_d, A.get_j_min_i().data(), num_rows * sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(A_d, A.get_values().data(), num_stripes * num_rows * sizeof(double), hipMemcpyHostToDevice));
    

    int num_threads = 1024;
    int num_blocks = 1;

    color_for_forward_pass_kernel<<<num_blocks, num_threads>>>(num_rows, num_stripes, diag_offset, A_d, j_min_i_d, colors_d);

    CHECK_CUDA(hipDeviceSynchronize());

    CHECK_CUDA(hipMemcpy(colors.data(), colors_d, num_rows * sizeof(int), hipMemcpyDeviceToHost));

    CHECK_CUDA(hipFree(colors_d));
    CHECK_CUDA(hipFree(j_min_i_d));
    CHECK_CUDA(hipFree(A_d));

    return colors;

}

std::vector <int> color_for_backward_pass(striped_Matrix <double> A){

    int num_rows = A.get_num_rows();
    int num_stripes = A.get_num_stripes();
    int diag_offset = A.get_diag_index();

    std::vector<int> colors(num_rows, -1);

    // put everything on the device
    int * colors_d;
    int * j_min_i_d;
    double * A_d;

    CHECK_CUDA(hipMalloc(&colors_d, num_rows * sizeof(int)));
    CHECK_CUDA(hipMalloc(&j_min_i_d, num_rows * sizeof(int)));
    CHECK_CUDA(hipMalloc(&A_d, num_stripes * num_rows * sizeof(double)));

    CHECK_CUDA(hipMemset(colors_d, -1, num_rows * sizeof(int)));
    CHECK_CUDA(hipMemcpy(j_min_i_d, A.get_j_min_i().data(), num_rows * sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(A_d, A.get_values().data(), num_stripes * num_rows * sizeof(double), hipMemcpyHostToDevice));
    

    int num_threads = 1024;
    int num_blocks = 1;

    color_for_backward_pass_kernel<<<num_blocks, num_threads>>>(num_rows, num_stripes, diag_offset, A_d, j_min_i_d, colors_d);

    CHECK_CUDA(hipDeviceSynchronize());

    CHECK_CUDA(hipMemcpy(colors.data(), colors_d, num_rows * sizeof(int), hipMemcpyDeviceToHost));

    CHECK_CUDA(hipFree(colors_d));
    CHECK_CUDA(hipFree(j_min_i_d));
    CHECK_CUDA(hipFree(A_d));

    return colors;

}