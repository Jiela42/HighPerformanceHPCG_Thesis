#include "hip/hip_runtime.h"
#include "HPCG_versions/striped_coloring.cuh"
#include "UtilLib/cuda_utils.hpp"
#include "UtilLib/utils.cuh"
#include "MatrixLib/coloring.cuh"
// #include <iostream>
// #include <hip/hip_runtime.h>

__global__ void striped_coloring_SymGS_forward_kernel(
    int color, int * colors,
    int num_rows, int num_cols,
    int num_stripes, int diag_offset,
    int * j_min_i,
    double * striped_A,
    double * x, double * y
){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int warp_id = tid / WARP_SIZE;
    int lane_id = tid % WARP_SIZE;
    int num_warps = blockDim.x * gridDim.x / WARP_SIZE;

    for(int i = warp_id; i < num_rows; i += num_warps){
        if(colors[i] == color){
            double my_sum = 0.0;
            for(int stripe = lane_id; stripe < num_stripes; stripe += WARP_SIZE){
                int col = j_min_i[stripe] + i;
                double val = striped_A[i * num_stripes + stripe];
                if(col < num_cols && col >= 0){
                    my_sum -= val * x[col];
                }
            }

            // reduce the my_sum using warp reduction
            for (int offset = WARP_SIZE/2; offset > 0; offset /= 2){
                my_sum += __shfl_down_sync(0xFFFFFFFF, my_sum, offset);
            }

            __syncthreads();
            if (lane_id == 0){
                double diag = striped_A[i * num_stripes + diag_offset];
                double sum = diag * x[i] + y[i] + my_sum;
                x[i] = sum / diag;           
            }
            __syncthreads();
        }
    }
}


__global__ void striped_coloring_SymGS_backward_kernel(
    // int color, int * colors,
    int num_rows, int num_cols,
    int num_stripes, int diag_offset,
    int * j_min_i,
    double * striped_A,
    double * x, double * y
){

    int lane = threadIdx.x % WARP_SIZE;
    __shared__ double shared_diag[1];

    for (int i = num_rows-1; i >= 0; i--){
        
        double my_sum = 0.0;
        for (int stripe = lane; stripe < num_stripes; stripe += WARP_SIZE){
            int col = j_min_i[stripe] + i;
            double val = striped_A[i * num_stripes + stripe];
            if (col < num_cols && col >= 0){
                my_sum -= val * x[col];
            }
            if(i == col){
                shared_diag[0] = val;
            }
        }

        // reduce the my_sum using warp reduction
        for (int offset = WARP_SIZE/2; offset > 0; offset /= 2){
            my_sum += __shfl_down_sync(0xFFFFFFFF, my_sum, offset);
        }

        __syncthreads();
        if (lane == 0){
            double diag = shared_diag[0];
            double sum = diag * x[i] + y[i] + my_sum;
            x[i] = sum / diag;           
        }
        __syncthreads();
    }
}

__global__ void compute_num_colors_per_row(int num_rows, int max_num_colors, int * colors, int * num_colors_per_row){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    for(int i = tid; i < max_num_colors; i += gridDim.x * blockDim.x){
        int num_rows_per_i = 0;
        for(int j = 0; j < num_rows; j++){
            if(colors[j] == i){
                num_rows_per_i++;
            }
        }
        num_colors_per_row[i] = num_rows_per_i;
    }

}

template <typename T>
void striped_coloring_Implementation<T>::striped_coloring_computeSymGS(
    striped_Matrix<T> & A, // we pass A for the metadata
    T * striped_A_d, // the data of matrix A is already on the device
    int num_rows, int num_cols,
    int num_stripes, // the number of stripes in the striped matrix
    int * j_min_i, // this is a mapping for calculating the j of some entry i,j in the striped matrix
    T * x_d, T * y_d // the vectors x and y are already on the device
){
    int diag_offset = A.get_diag_index();
    assert(num_stripes == A.get_num_stripes());
    assert(num_rows == A.get_num_rows());
    assert(num_cols == A.get_num_cols());
    assert(diag_offset >= 0);

    // first we need to color the matrix
    // we make a device vector for the colors
    int * colors_d;

    // we allocate the memory for the colors
    CHECK_CUDA(hipMalloc(&colors_d, num_rows * sizeof(int)));

    // we initialize the colors to -1
    CHECK_CUDA(hipMemset(colors_d, -1, num_rows * sizeof(int)));

    color_for_forward_pass_kernel<<<1, 1024>>>(num_rows, num_stripes, diag_offset, striped_A_d, j_min_i, colors_d);

    CHECK_CUDA(hipDeviceSynchronize());

    int max_color;
    CHECK_CUDA(hipMemcpy(&max_color, &colors_d[num_rows-1], sizeof(int), hipMemcpyDeviceToHost));

    std::vector<int> num_colors_per_row(max_color, -1);
    // int * num_colors_per_row_d;
    // CHECK_CUDA(hipMalloc(&num_colors_per_row_d, max_color * sizeof(int)));

    // int num_blocks = std::min(ceiling_division(max_color, 1024));
    // compute_num_colors_per_row<<<num_blocks, 1024>>>(num_rows, max_color, colors_d, num_colors_per_row_d);

    // CHECK_CUDA(hipDeviceSynchronize());
    // CHECK_CUDA(hipMemcpy(num_colors_per_row.data(), num_colors_per_row_d, max_color * sizeof(int), hipMemcpyDeviceToHost));

    for(int color = 0; color < max_color; color++){
        // we need to do a forward pass
        int num_blocks = std::min(ceiling_division(num_rows, 1024/WARP_SIZE), MAX_NUM_BLOCKS);
        striped_coloring_SymGS_forward_kernel<<<num_blocks, 1024>>>(
        color, colors_d,
        num_rows, num_cols,
        num_stripes, diag_offset,
        j_min_i,
        striped_A_d,
        x_d, y_d
        );
        CHECK_CUDA(hipDeviceSynchronize());
    }

    // we need to do a backward pass
    int num_blocks = 1;
    int num_threads = WARP_SIZE;
    striped_coloring_SymGS_backward_kernel<<<num_blocks, num_threads>>>(
        num_rows, num_cols,
        num_stripes, diag_offset,
        j_min_i,
        striped_A_d,
        x_d, y_d
    );

    CHECK_CUDA(hipDeviceSynchronize());

}

// explicit template instantiation
template class striped_coloring_Implementation<double>;