#include "hip/hip_runtime.h"
#include "HPCG_versions/striped_coloring.cuh"
#include "UtilLib/cuda_utils.hpp"
#include "UtilLib/utils.cuh"
#include "MatrixLib/coloring.cuh"
// #include <iostream>
// #include <hip/hip_runtime.h>

__global__ void striped_coloring_half_SymGS_kernel(
    int color, int * color_pointer, int * color_sorted_rows,
    int num_rows, int num_cols,
    int num_stripes, int diag_offset,
    int * j_min_i,
    double * striped_A,
    double * x, double * y
){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int warp_id = tid / WARP_SIZE;
    int lane_id = tid % WARP_SIZE;
    int num_warps = blockDim.x * gridDim.x / WARP_SIZE;

    int start = color_pointer[color];
    int end = color_pointer[color+1];

    for(int i = warp_id + start; i < end; i += num_warps){
        int row = color_sorted_rows[i];
            double my_sum = 0.0;
            for(int stripe = lane_id; stripe < num_stripes; stripe += WARP_SIZE){
                int col = j_min_i[stripe] + row;
                double val = striped_A[row * num_stripes + stripe];
                if(col < num_cols && col >= 0){
                    my_sum -= val * x[col];
                }
            }

            // reduce the my_sum using warp reduction
            for (int offset = WARP_SIZE/2; offset > 0; offset /= 2){
                my_sum += __shfl_down_sync(0xFFFFFFFF, my_sum, offset);
            }

            __syncthreads();
            if (lane_id == 0){
                double diag = striped_A[row * num_stripes + diag_offset];
                double sum = diag * x[row] + y[row] + my_sum;
                x[row] = sum / diag;           
            }
            __syncthreads();
        
    }
}


__global__ void striped_coloring_SymGS_backward_kernel(
    int color, int * colors,
    int num_rows, int num_cols,
    int num_stripes, int diag_offset,
    int * j_min_i,
    double * striped_A,
    double * x, double * y
){

     int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int warp_id = tid / WARP_SIZE;
    int lane_id = tid % WARP_SIZE;
    int num_warps = blockDim.x * gridDim.x / WARP_SIZE;

    for(int i = num_rows-1-warp_id; i >= 0; i += num_warps){
        if(colors[i] == color){
            double my_sum = 0.0;
            for(int stripe = lane_id; stripe < num_stripes; stripe += WARP_SIZE){
                int col = j_min_i[stripe] + i;
                double val = striped_A[i * num_stripes + stripe];
                if(col < num_cols && col >= 0){
                    my_sum -= val * x[col];
                }
            }

            // reduce the my_sum using warp reduction
            for (int offset = WARP_SIZE/2; offset > 0; offset /= 2){
                my_sum += __shfl_down_sync(0xFFFFFFFF, my_sum, offset);
            }

            __syncthreads();
            if (lane_id == 0){
                double diag = striped_A[i * num_stripes + diag_offset];
                double sum = diag * x[i] + y[i] + my_sum;
                x[i] = sum / diag;           
            }
            __syncthreads();
        }
    }
}

__global__ void compute_num_colors_per_row(int num_rows, int max_num_colors, int * colors, int * num_colors_per_row){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    for(int i = tid; i < max_num_colors; i += gridDim.x * blockDim.x){
        int num_rows_per_i = 0;
        for(int j = 0; j < num_rows; j++){
            if(colors[j] == i){
                num_rows_per_i++;
            }
        }
        num_colors_per_row[i] = num_rows_per_i;
    }
}

template <typename T>
void striped_coloring_Implementation<T>::striped_coloring_computeSymGS(
    striped_Matrix<T> & A,
    T * x_d, T * y_d // the vectors x and y are already on the device
){
    int diag_offset = A.get_diag_index();

    int num_rows = A.get_num_rows();
    int num_cols = A.get_num_cols();
    int num_stripes = A.get_num_stripes();
    int * j_min_i = A.get_j_min_i_d();
    T * striped_A_d = A.get_values_d();

    int nx = A.get_nx();
    int ny = A.get_ny();
    int nz = A.get_nz();

    // first we need to color the matrix
    // we make a device vector for the colors
    int * color_pointer_d;
    int * color_sorted_rows_d;

    // we allocate the memory for the colors
    CHECK_CUDA(hipMalloc(&color_pointer_d, (num_rows+1) * sizeof(int)));
    CHECK_CUDA(hipMalloc(&color_sorted_rows_d, num_rows * sizeof(int)));
    
    CHECK_CUDA(hipMemset(color_pointer_d, 0, (num_rows+1) * sizeof(int)));

    // we need to get the colors
    get_color_row_mapping(nx, ny, nz, color_pointer_d, color_sorted_rows_d);

    // the number of blocks is now dependent on the maximum number of rows per color

    int max_num_rows_per_color = std::min(nx * ny / 4, std::min(nx * nz / 2, ny * nz));
    int max_color = (nx-1) + 2 * (ny-1) + 4 * (nz-1);

    int num_blocks = std::min(ceiling_division(max_num_rows_per_color, 1024/WARP_SIZE), MAX_NUM_BLOCKS);
    for(int color = 0; color <= max_color; color++){
        // we need to do a forward pass
        striped_coloring_half_SymGS_kernel<<<num_blocks, 1024>>>(
        color, color_pointer_d, color_sorted_rows_d,
        num_rows, num_cols,
        num_stripes, diag_offset,
        j_min_i,
        striped_A_d,
        x_d, y_d
        );
        CHECK_CUDA(hipDeviceSynchronize());
    }

    // we need to do a backward pass,
    // the colors for this are the same just in reverse order
    
    for(int color = max_color; color  >= 0; color--){

        striped_coloring_half_SymGS_kernel<<<num_blocks, 1024>>>(
        color, color_pointer_d, color_sorted_rows_d,
        num_rows, num_cols,
        num_stripes, diag_offset,
        j_min_i,
        striped_A_d,
        x_d, y_d
        );
        CHECK_CUDA(hipDeviceSynchronize());
    }

    // free the memory
    CHECK_CUDA(hipFree(color_pointer_d));
    CHECK_CUDA(hipFree(color_sorted_rows_d));
    
}

// explicit template instantiation
template class striped_coloring_Implementation<double>;