#include "hip/hip_runtime.h"
#include "HPCG_versions/banded_warp_reduction.cuh"
#include "UtilLib/cuda_utils.hpp"
#include <iostream>

__inline__ __device__ void loop_body(int lane, int i, int num_cols, int num_bands, int * j_min_i, double * banded_A, double * x, double * y, double * shared_diag){
    
    double my_sum = 0.0;
    for (int band = lane; band < num_bands; band += WARP_SIZE){
        int col = j_min_i[band] + i;
        double val = banded_A[i * num_bands + band];
        if (col < num_cols && col >= 0){
            my_sum -= val * x[col];
        }
        if(i == col){
            shared_diag[0] = val;
        }
    }

    // reduce the my_sum using warp reduction
    for (int offset = WARP_SIZE/2; offset > 0; offset /= 2){
        my_sum += __shfl_down_sync(0xFFFFFFFF, my_sum, offset);
    }

    __syncthreads();
    if (lane == 0){
        double diag = shared_diag[0];
        double sum = diag * x[i] + y[i] + my_sum;
        x[i] = sum / diag;           
    }
    __syncthreads();
}

__global__ void banded_warp_reduction_SymGS_kernel(
    int num_rows, int num_cols,
    int num_bands,
    int * j_min_i,
    double * banded_A,
    double * x, double * y
){
    // note that here x is the result vector and y is the input vector

    __shared__ double diag_value[1];
    int lane = threadIdx.x % WARP_SIZE;
    
    // forward pass
    for (int i = 0; i < num_rows; i++){
        loop_body(lane, i, num_cols, num_bands, j_min_i, banded_A, x, y, diag_value);
    }

    __syncthreads();

    // backward pass
    for (int i = num_rows-1; i >= 0; i--){
        loop_body(lane, i, num_cols, num_bands, j_min_i, banded_A, x, y, diag_value);
    }
}

template <typename T>
void banded_warp_reduction_Implementation<T>::banded_warp_reduction_computeSymGS(
    banded_Matrix<T> & A, // we pass A for the metadata
    T * banded_A_d, // the data of matrix A is already on the device
    int num_rows, int num_cols,
    int num_bands, // the number of bands in the banded matrix
    int * j_min_i, // this is a mapping for calculating the j of some entry i,j in the banded matrix
    T * x_d, T * y_d // the vectors x and y are already on the device
        
){

    assert(num_rows == A.get_num_rows());
    assert(num_cols == A.get_num_cols());
    
    // because this is sequential, we only spawn one warp
    int num_threads = WARP_SIZE;
    int num_blocks = 1;

    banded_warp_reduction_SymGS_kernel<<<num_blocks, num_threads>>>(
        num_rows, num_cols,
        num_bands,
        j_min_i,
        banded_A_d,
        x_d, y_d
    );

    CHECK_CUDA(hipDeviceSynchronize());
   
}

// Explicit instantiation of the template
template class banded_warp_reduction_Implementation<double>;