#include "hip/hip_runtime.h"
#include "HPCG_versions/striped_multi_GPU.cuh"
#include "UtilLib/cuda_utils.hpp"
#include "UtilLib/utils.cuh"
#include "UtilLib/hpcg_mpi_utils.cuh"
// #include "MatrixLib/coloring.cuh"
// #include <iostream>
// #include <hip/hip_runtime.h>

__inline__ __device__ global_int_t local_i_to_global_i(
    int i, 
    int nx, int ny, int nz, 
    global_int_t gnx, global_int_t gny, global_int_t gnz,
    global_int_t gi0
    )
    {
        int local_i_x = i % nx;
        int local_i_y = (i % (nx * ny)) / nx;
        int local_i_z = i / (nx * ny);
        return gi0 + local_i_x + local_i_y * gnx + local_i_z * (gnx * gny);
}

__inline__ __device__ local_int_t global_i_to_halo_i(
    int i,
    int nx, int ny, int nz,
    global_int_t gnx, global_int_t gny, global_int_t gnz,
    global_int_t gi0,
    int px, int py, int pz
    )
    {
        local_int_t global_j_x = i % gnx;
        local_int_t global_j_y = (i % (gnx * gny)) / gnx;
        local_int_t global_j_z = i / (gnx * gny);
        int halo_j_x = global_j_x - px * nx + 1;
        int halo_j_y = global_j_y - py * ny + 1;
        int halo_j_z = global_j_z - pz * nz + 1;
        return halo_j_x + halo_j_y * (nx+2) + halo_j_z * ((nx+2) * (ny+2));
}

__global__ void striped_box_coloring_half_SymGS_kernel(
    int cooperation_number,
    int color, int bx, int by, int bz,
    int nx, int ny, int nz,
    int num_rows, int num_cols,
    int num_stripes, int diag_offset,
    int * j_min_i,
    double * striped_A,
    DataType * x, DataType * y,
    global_int_t gnx, global_int_t gny, global_int_t gnz,
    global_int_t gi0,
    int px, int py, int pz
){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int coop_group_id = tid / cooperation_number;
    int lane = tid % cooperation_number;
    int num_coop_groups = blockDim.x * gridDim.x / cooperation_number;

    // now we do some calculations concerning the colors
    // cols are in the x direction
    // rows are in the y direction
    // stripes are in the z direction

    int num_color_cols = nx / bx;
    int num_color_rows = ny / by;
    int num_color_faces = nz / bz;

    // How is the vector colored
    int color_offs_x = color % bx; //gives x-xcoordinate of first appearance of color
    int color_offs_y = (color - color_offs_x) % (bx * by) / bx; //gives y-coordinate of first appearance of color
    int color_offs_z = (color - color_offs_x - bx * color_offs_y) / (bx * by); //gives z-coordinate of first appearance of color

    num_color_cols = (color_offs_x < nx % bx) ? (num_color_cols + 1) : num_color_cols;
    num_color_rows = (color_offs_y < ny % by) ? (num_color_rows + 1) : num_color_rows;
    num_color_faces = (color_offs_z < nz % bz) ? (num_color_faces + 1) : num_color_faces;

    int num_nodes_with_color = num_color_cols * num_color_rows * num_color_faces;

    for (int i = coop_group_id; i < num_nodes_with_color; i += num_coop_groups){
        
        // find out the position of the node (only considering faces, cols and rows that actually have that color)
        int ix = i % num_color_cols;
        int iy = ((i % (num_color_cols * num_color_rows))) / num_color_cols;
        int iz = i / (num_color_cols * num_color_rows);
        
        // adjust the counter to the correct position when all nodes are considered
        ix = ix * bx + color_offs_x;
        iy = iy * by + color_offs_y;
        iz = iz * bz + color_offs_z;

        //compute the local index of the node and convert to global index
        local_int_t li = ix + iy * nx + iz * nx * ny;

        global_int_t gi = local_i_to_global_i(li, nx, ny, nz, gnx, gny, gnz, gi0);
        DataType my_sum = 0.0;
        for(int stripe = lane; stripe < num_stripes; stripe += cooperation_number){
            global_int_t gj = j_min_i[stripe] + gi;
            if (gj>= 0 && gj < gnx * gny * gnz) {
                //convert gj to halo coordinate hj which is the memory location of gj in the halo struct
                local_int_t hj =  global_i_to_halo_i(gj, nx, ny, nz, gnx, gny, gnz, gi0, px, py, pz);
                my_sum -= striped_A[li * num_stripes + stripe] * x[hj];
            }
        }

        // reduce the my_sum using warp reduction
        for (int offset = cooperation_number/2; offset > 0; offset /= 2){
            my_sum += __shfl_down_sync(0xFFFFFFFF, my_sum, offset);
        }

        __syncthreads();
        if (lane == 0){
            local_int_t hi =  global_i_to_halo_i(gi, nx, ny, nz, gnx, gny, gnz, gi0, px, py, pz);
            DataType diag = striped_A[li * num_stripes + diag_offset];
            DataType sum = diag * x[hi] + y[hi] + my_sum;
            x[hi] = sum / diag;           
        }
        __syncthreads();
    }
}

/*
__global__ void striped_box_coloring_half_SymGS_kernel(
    int cooperation_number,
    int color, int bx, int by, int bz,
    int nx, int ny, int nz,
    int num_rows, int num_cols,
    int num_stripes, int diag_offset,
    int * j_min_i,
    double * striped_A,
    double * x, double * y,
    global_int_t gnx, global_int_t gny, global_int_t gnz,
    global_int_t gi0,
    int px, int py, int pz
){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int coop_group_id = tid / cooperation_number;
    int lane = tid % cooperation_number;
    int num_coop_groups = blockDim.x * gridDim.x / cooperation_number;

    // How is the vector colored
    int color_offs_x = color % bx; //gives x-xcoordinate of first appearance of color
    int color_offs_y = (color - color_offs_x) % (bx * by) / bx; //gives y-coordinate of first appearance of color
    int color_offs_z = (color - color_offs_x - bx * color_offs_y) / (bx * by); //gives z-coordinate of first appearance of color

    //first appearance of that color in local space, we need to adjust for the fact that nx, ny, nz might not be divisible by bx, by, bz and hence the first appearance of the color might not be at 0, 0, 0 locally
    global_int_t cx0 = (color_offs_x + bx - (nx % bx)) % bx;
    global_int_t cy0 = (color_offs_y + by - (ny % by)) % by;
    global_int_t cz0 = (color_offs_z + bz - (nz % bz)) % bz;

    //how often does this color appear in each direction
    int num_color_in_x = 1 + (nx - cx0) / bx;
    int num_color_in_y = 1 + (ny - cy0) / by;
    int num_color_in_z = 1 + (nz - cz0) / bz;
    int num_nodes_with_color = num_color_in_x * num_color_in_y * num_color_in_z;

    for (int i = coop_group_id; i < num_nodes_with_color; i += num_coop_groups){
        
        // find out the position of the node based on the number of nodes with this color in each direction
        int iz = i % num_color_in_x;
        int iy = (i % (num_color_in_x * num_color_in_y)) / num_color_in_x;
        int ix = i / (num_color_in_x * num_color_in_y);
        
        // adjust the counter to the correct position when all nodes are considered
        ix = cx0 + ix * bx;
        iy = cy0 + iy * by;
        iz = cz0 + iz * bz;
        //guard against out of bounds
        if(ix >= nx || iy >= ny || iz >= nz || ix < 0 || iy < 0 || iz < 0){
            continue;
        }

        //compute the local index of the node and convert to global index
        local_int_t li = ix * ny * nz + iy * nz + iz;
        global_int_t gi = local_i_to_global_i(li, nx, ny, nz, gnx, gny, gnz, gi0);
        DataType my_sum = 0.0;
        for(int stripe = lane; stripe < num_stripes; stripe += cooperation_number){
            global_int_t gj = j_min_i[stripe] + gi;
            if (gj >= 0 && gj < gnx * gny * gnz) {
                //convert gj to halo coordinate hj which is the memory location of gj in the halo struct
                local_int_t hj =  global_i_to_halo_i(gj, nx, ny, nz, gnx, gny, gnz, gi0, px, py, pz);
                my_sum -= striped_A[li * num_stripes + stripe] * x[hj];
            }
        }

        // reduce the my_sum using warp reduction
        for (int offset = cooperation_number/2; offset > 0; offset /= 2){
            my_sum += __shfl_down_sync(0xFFFFFFFF, my_sum, offset);
        }

        __syncthreads();
        if (lane == 0){
            local_int_t hi =  global_i_to_halo_i(gi, nx, ny, nz, gnx, gny, gnz, gi0, px, py, pz);
            double diag = striped_A[li * num_stripes + diag_offset];
            double sum = diag * x[hi] + y[hi] + my_sum;
            x[hi] = sum / diag;           
        }
        __syncthreads();
    }
}
*/

template <typename T>
void striped_multi_GPU_Implementation<T>::striped_box_coloring_multi_GPU_computeSymGS(
    striped_Matrix<T> & A,
    Halo *x_d, Halo *b_d, // the vectors x and y are already on the device
    Problem *problem,
    int *j_min_i
){
    //some geometrical information
    int nx = problem->nx;
    int ny = problem->ny;
    int nz = problem->nz;
    assert(nx % 3 == 0);
    assert(ny % 3 == 0);
    assert(nz % 3 == 0);
    global_int_t gnx = problem->gnx;
    global_int_t gny = problem->gny;
    global_int_t gnz = problem->gnz;
    global_int_t gi0 = problem->gi0;
    int px = problem->px;
    int py = problem->py;
    int pz = problem->pz;
    
    int diag_offset = A.get_diag_index();

    int num_rows = A.get_num_rows();
    int num_cols = A.get_num_cols();
    int num_stripes = A.get_num_stripes();
    DataType * striped_A_d = A.get_values_d();

    assert(diag_offset >= 0);

    // check that the box size does not violate dependencies
    // we assume a 3d 27pt stencil
    int bx = this->bx;
    int by = this->by;
    int bz = this->bz;

    assert(bx >= 3);
    assert(by >= 3);
    assert(bz >= 3);

    // std::cout << "bx: " << bx << " by: " << by << " bz: " << bz << std::endl;
    /*
    int max_iterations = this->max_SymGS_iterations;
    double threshold_rr_Norm = 1.0;

    if(this->norm_based and max_iterations > 1){
        threshold_rr_Norm = this->getSymGS_rrNorm_zero_init(nx, ny, nz);
        assert(threshold_rr_Norm >= 0.0);
    }
    */

    int cooperation_number = this->SymGS_cooperation_number;

    // the number of blocks is now dependent on the maximum number of rows per color
    int num_colors = bx * by * bz;
    int max_color =  num_colors - 1;
    // std::cout << "max_color: " << max_color << std::endl;
    int max_num_rows_per_color = ceiling_division(nx, bx) * ceiling_division(ny, by) * ceiling_division(nz, bz);

    int num_blocks = std::min(ceiling_division(max_num_rows_per_color, 1024/cooperation_number), MAX_NUM_BLOCKS);
    /*
    double L2_norm_y;

    hipStream_t y_Norm_stream;
    CHECK_CUDA(hipStreamCreate(&y_Norm_stream));

    printf("ALERT: L2_norm_for_Device_Vector is not implemented for multi GPU!  Result is going to be wrong!");
    //L2_norm_for_Device_Vector(y_Norm_stream, num_rows, b_d, &L2_norm_y);
    */
    // to do the L2 norm asynchroneously we do the first iteration outside of the loop
    for(int color = 0; color < max_color; color++){
            // we need to do a forward pass
            striped_box_coloring_half_SymGS_kernel<<<num_blocks, 1024>>>(
            cooperation_number,
            color, bx, by, bz,
            nx, ny, nz,
            num_rows, num_cols,
            num_stripes, diag_offset,
            j_min_i,
            striped_A_d,
            x_d->x_d, b_d->x_d,
            gnx, gny, gnz, gi0, px, py, pz
            );
            CHECK_CUDA(hipDeviceSynchronize());
            ExchangeHalo(x_d, problem);
        }

    // we need to do a backward pass,
    // the colors for this are the same just in reverse order
    
    for(int color = max_color; color  >= 0; color--){
        striped_box_coloring_half_SymGS_kernel<<<num_blocks, 1024>>>(
        cooperation_number,
        color, bx, by, bz,
        nx, ny, nz,
        num_rows, num_cols,
        num_stripes, diag_offset,
        j_min_i,
        striped_A_d,
        x_d->x_d, b_d->x_d,
        gnx, gny, gnz, gi0, px, py, pz
        );
        CHECK_CUDA(hipDeviceSynchronize());
        ExchangeHalo(x_d, problem);
    }
    /*
    printf("ALERT: L2_norm_ is not implemented for multi GPU! Result is going to be wrong!");
    double L2_norm = 1.0; //L2_norm_for_SymGS(A, x_d, y_d);
    CHECK_CUDA(hipStreamSynchronize(y_Norm_stream));
    CHECK_CUDA(hipStreamDestroy(y_Norm_stream));

    double rr_norm = L2_norm / L2_norm_y;

    // std::cout << "rr_norm after one iteration: " << rr_norm << std::endl;

    int iter = 1;


    // this while loop only kicks in if we are benchmarking or testing SymGS itself
    // as a part of MG or CG we will not use this loop since in this case SymGS is only executed once
    while (iter < max_iterations and rr_norm > threshold_rr_Norm){

        for(int color = 0; color <= max_color; color++){
            // we need to do a forward pass
            striped_box_coloring_half_SymGS_kernel<<<num_blocks, 1024>>>(
            cooperation_number,
            color, bx, by, bz,
            nx, ny, nz,
            num_rows, num_cols,
            num_stripes, diag_offset,
            j_min_i,
            striped_A_d,
            x_d->x_d, b_d->x_d,
            gnx, gny, gnz, gi0, px, py, pz
            );
            CHECK_CUDA(hipDeviceSynchronize());
            ExchangeHalo(x_d, problem);
        }

        // we need to do a backward pass,
        // the colors for this are the same just in reverse order
        
        for(int color = max_color; color  >= 0; color--){

            striped_box_coloring_half_SymGS_kernel<<<num_blocks, 1024>>>(
            cooperation_number,
            color, bx, by, bz,
            nx, ny, nz,
            num_rows, num_cols,
            num_stripes, diag_offset,
            j_min_i,
            striped_A_d,
            x_d->x_d, b_d->x_d,
            gnx, gny, gnz, gi0, px, py, pz
            );
            CHECK_CUDA(hipDeviceSynchronize());
            ExchangeHalo(x_d, problem);
        }

        printf("ALERT: L2_norm_ is not implemented for multi GPU! Result is going to be wrong!");
        double L2_norm = 1.0; //L2_norm_for_SymGS(A, x_d, y_d);
   
        rr_norm = L2_norm / L2_norm_y;

        iter ++;
    }*/

    // std::cout << "SymGS for size " << nx << "x" << ny << "x" << nz << " took " << iter << " iterations." << std::endl;
    // std::cout << "RR norm after " << iter << " iterations: " << rr_norm << std::endl;
    // std::cout << "Threshold RR norm: " << threshold_rr_Norm << std::endl;
}

// explicit template instantiation
template class striped_multi_GPU_Implementation<DataType>;