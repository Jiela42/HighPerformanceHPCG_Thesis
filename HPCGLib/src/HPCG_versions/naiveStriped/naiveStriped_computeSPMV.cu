#include "hip/hip_runtime.h"
#include "HPCG_versions/naiveStriped.cuh"
#include "UtilLib/utils.cuh"
#include <hip/hip_runtime.h>


__global__ void naiveStriped_SPMV_kernel(
        DataType* striped_A,
        local_int_t num_rows, int num_stripes, local_int_t * j_min_i,
        DataType* x, DataType* y
    )
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // every thread computes one or more rows of the matrix
    for (local_int_t i = tid; i < num_rows; i += blockDim.x * gridDim.x) {
        // compute the matrix-vector product for the ith row
        double sum_i = 0;
        for (int stripe = 0; stripe < num_stripes; stripe++) {
            local_int_t j = i + j_min_i[stripe];
            local_int_t current_row = i * num_stripes;
            if (j >= 0 && j < num_rows) {
                sum_i += striped_A[current_row + stripe] * x[j];
            }
        }
        y[i] = sum_i;
    }
}
    

template <typename T>
void naiveStriped_Implementation<T>::naiveStriped_computeSPMV(
        striped_Matrix<T>& A,
        T * x_d, T * y_d // the vectors x and y are already on the device
    ) {

        local_int_t num_rows = A.get_num_rows();
        int num_stripes = A.get_num_stripes();
        local_int_t * j_min_i = A.get_j_min_i_d();
        T * striped_A_d = A.get_values_d();

        // call the kernel for the naive striped SPMV
        // since every thread is working on one or more rows we need to base the number of threads on that
        int num_threads = NUM_CORES_PER_SM * 4;
        int num_blocks = std::min(NUM_PHYSICAL_CORES, ceiling_division(num_rows, num_threads));

        // call the kernel
        naiveStriped_SPMV_kernel<<<num_blocks, num_threads>>>(
            striped_A_d, num_rows, num_stripes, j_min_i, x_d, y_d
        );

        // synchronize the device
        hipDeviceSynchronize();
    }

// explicit template instantiation
template class naiveStriped_Implementation<double>;