#include "hip/hip_runtime.h"
#include "HPCG_versions/cusparse.hpp"
#include "UtilLib/cuda_utils.hpp"
#include "UtilLib/utils.cuh"

#include <thrust/device_vector.h>
#include <thrust/reduce.h>
#include <thrust/functional.h>
#include <iostream>
#include <hipblas.h>

__global__ void elem_wise_mult_of_vectors_kernel(int num_rows, double *x, double *y, double *z){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    for(int row = tid; row < num_rows; row += blockDim.x * gridDim.x){
        z[row] = x[row] * y[row];
    }
}

template <typename T>
void cuSparse_Implementation<T>::cusparse_computeDot(
    sparse_CSR_Matrix<T>& A, //we only pass A for the metadata
    T * x_d,
    T * y_d,
    T * z_d
    ){
    // we compute z = xy

    int num_rows = A.get_num_rows();

    // // use cuBLAS to compute the dot product
    // hipblasHandle_t handle;
    // hipblasCreate(&handle);

    // int incx, incy = 1;
    // double result_host = 0.0;

    // CHECK_CUBLAS(hipblasDdot(handle, num_rows, x_d, incx, y_d, incy, &result_host));

    // CHECK_CUDA(hipDeviceSynchronize());

    // printf("result_host: %f\n", result_host);

    // CHECK_CUDA(hipMemcpy(z_d, &result_host, sizeof(T), hipMemcpyHostToDevice));

    // hipblasDestroy(handle);


    // allocate memory for the intermediate vector
    double *intermediate_vector;
    CHECK_CUDA(hipMalloc(&intermediate_vector, num_rows * sizeof(double)));
    // std::cout << "we do run the expected implementation" << std::endl;

    int threads = 1024;
    int num_blocks = min(ceiling_division(num_rows, threads), MAX_NUM_BLOCKS);

    elem_wise_mult_of_vectors_kernel<<<num_blocks, threads>>>(num_rows, x_d, y_d, intermediate_vector);

    CHECK_CUDA(hipDeviceSynchronize());

    // use thrust to reduce
    thrust::device_ptr<double> thrust_intermediate_vector(intermediate_vector);
    double result = thrust::reduce(thrust_intermediate_vector, thrust_intermediate_vector + num_rows, 0.0, thrust::plus<double>());

    // write the result to the device
    CHECK_CUDA(hipMemcpy(z_d, &result, sizeof(double), hipMemcpyHostToDevice));

}

template class cuSparse_Implementation<double>;