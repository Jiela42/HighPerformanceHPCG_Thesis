#include "hip/hip_runtime.h"
#include "HPCG_versions/striped_coloringPrecomputed.cuh"
#include "HPCG_versions/striped_coloring.cuh"
#include "UtilLib/cuda_utils.hpp"
#include "UtilLib/utils.cuh"
#include "MatrixLib/coloring.cuh"


// note: this function uses the same kernel as the implementation striped_coloring (hence the import)
// the difference is that this version does not need to compute the coloring, it is already precomputed
template <typename T>
void striped_coloringPrecomputed_Implementation<T>::striped_coloringPrecomputed_computeSymGS(
    striped_Matrix<T> & A,
    T * x_d, T * y_d // the vectors x and y are already on the device
){
    int diag_offset = A.get_diag_index();

    local_int_t num_rows = A.get_num_rows();
    local_int_t num_cols = A.get_num_cols();
    int num_stripes = A.get_num_stripes();
    local_int_t * j_min_i = A.get_j_min_i_d();
    T * striped_A_d = A.get_values_d();

    // the coloring was already computed, we can grab the pointers from the striped matrix object
    local_int_t * color_pointer_d = A.get_color_pointer_d();
    local_int_t * color_sorted_rows_d = A.get_color_sorted_rows_d();

    assert(diag_offset >= 0);
    // this assertion is here such that we don't benchmark the coloring computation
    // usually if these pointers are null we can just call the generate coloring function on the matrix
    assert(color_pointer_d != nullptr);
    assert(color_sorted_rows_d != nullptr);
    
    int nx = A.get_nx();
    int ny = A.get_ny();
    int nz = A.get_nz();

    // the number of blocks is now dependent on the maximum number of rows per color

    int max_num_rows_per_color = std::min(nx * ny / 4, std::min(nx * nz / 2, ny * nz));
    int max_color = (nx-1) + 2 * (ny-1) + 4 * (nz-1);

    int num_blocks = std::min(ceiling_division(max_num_rows_per_color, 1024/WARP_SIZE), MAX_NUM_BLOCKS);
    
    int max_iterations = this->max_SymGS_iterations;
    // std::cout << "max_iterations = " << max_iterations << std::endl;
    double norm0 = 1.0;
    double normi = norm0;

    if(max_iterations != 1){
        // compute the original L2 norm
        norm0 = this->L2_norm_for_SymGS(A, x_d, y_d);
    }

    for(int i = 0; i < max_iterations && normi/norm0 > this->SymGS_tolerance; i++){
        for(int color = 0; color <= max_color; color++){
            // we need to do a forward pass
            striped_coloring_half_SymGS_kernel<<<num_blocks, 1024>>>(
            color, color_pointer_d, color_sorted_rows_d,
            num_rows, num_cols,
            num_stripes, diag_offset,
            j_min_i,
            striped_A_d,
            x_d, y_d
            );
            CHECK_CUDA(hipDeviceSynchronize());
        }
    
        // we need to do a backward pass,
        // the colors for this are the same just in reverse order
        
        for(int color = max_color; color  >= 0; color--){
    
            striped_coloring_half_SymGS_kernel<<<num_blocks, 1024>>>(
            color, color_pointer_d, color_sorted_rows_d,
            num_rows, num_cols,
            num_stripes, diag_offset,
            j_min_i,
            striped_A_d,
            x_d, y_d
            );
            CHECK_CUDA(hipDeviceSynchronize());
        }

        if(max_iterations != 1){
            normi = this->L2_norm_for_SymGS(A, x_d, y_d);
        }
    }
    
}

// explicit template instantiation
template class striped_coloringPrecomputed_Implementation<DataType>;