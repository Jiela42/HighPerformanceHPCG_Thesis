#include "hip/hip_runtime.h"



#include "HPCG_versions/striped_warp_reduction.cuh"
#include "UtilLib/cuda_utils.hpp"
#include <iostream>
#include <thrust/reduce.h>
#include <thrust/functional.h>
#include <thrust/device_ptr.h>

__global__ void reduce_sums(double * array, int num_elements, double * result_d){

    __shared__ double intermediate_sums[32];
    
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int lane = threadIdx.x % 32;
    int warp_id = threadIdx.x / 32;

    double my_sum = 0.0;

    for (int i = tid; i < num_elements; i += blockDim.x * gridDim.x){
        my_sum += array[i];
    }

    for (int offset = 16; offset > 0; offset /= 2){
        my_sum += __shfl_down_sync(0xFFFFFFFF, my_sum, offset);
    }

    __syncthreads();

    if (lane == 0){
        intermediate_sums[warp_id] = my_sum;
    }

    __syncthreads();

    if(warp_id == 0){
        my_sum = intermediate_sums[lane];
        for (int offset = 16; offset > 0; offset /= 2){
            my_sum += __shfl_down_sync(0xFFFFFFFF, my_sum, offset);
        }
    }

    __syncthreads();

    if(tid == 0){
        *result_d = my_sum;
    }
}

__global__ void striped_warp_reduction_dot_kernel(
    int num_rows,
    double * x_d,
    double * y_d,
    double * result_d
){

    __shared__ double intermediate_sums[32];

    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int lane = threadIdx.x % 32;
    // warp_id within the block
    int warp_id = threadIdx.x / 32;

    // first we reduce as much as we can without cooperation
    double my_sum = 0.0;

    for (int i = tid; i < num_rows; i += blockDim.x * gridDim.x){
        // if (y_d[i] != 0.0){
        //     printf("y_d[%d] = %f\n", i, y_d[i]);
        // }
        my_sum += x_d[i] * y_d[i];
        // printf("i = %d, x_d[i] = %f, y_d[i] = %f\n", i, x_d[i], y_d[i]);
    }

    // now we cooperatively reduce the sum

    for (int offset = 16; offset > 0; offset /= 2){
        my_sum += __shfl_down_sync(0xFFFFFFFF, my_sum, offset);
    }

    __syncthreads();

    if (lane == 0){
        intermediate_sums[warp_id] = my_sum;
    }

    __syncthreads();

    // now we reduce the intermediate sums
    if (threadIdx.x < 32){
        my_sum = intermediate_sums[threadIdx.x];
        for (int offset = 16; offset > 0; offset /= 2){
            my_sum += __shfl_down_sync(0xFFFFFFFF, my_sum, offset);
        }
    }

    __syncthreads();

    // printf("my_sum = %f\n", my_sum);
    if(threadIdx.x == 0){
        result_d[blockIdx.x] = my_sum;
        // if (my_sum != 0.0){

        // printf("result_d[%d] = %f\n", blockIdx.x, result_d[blockIdx.x]);
        // }
    }
}

template <typename T>
void striped_warp_reduction_Implementation<T>::striped_warp_reduction_computeDot(
    striped_Matrix<T>& A, //we only pass A for the metadata
    T * x_d,
    T * y_d,
    T * result_d
    ){
    
    int coop_num = this->dot_cooperation_number;
    // we compute z = xy

    int num_rows = A.get_num_rows();
    int num_threads = 1024;
    int num_blocks = std::min(num_rows/(num_threads*coop_num), MAX_NUM_BLOCKS);
    // we need at least one block
    num_blocks = max(num_blocks, 1);

    // allocate memory for the intermediate vector
    double *intermediate_sums_d;

    CHECK_CUDA(hipMalloc(&intermediate_sums_d, num_blocks * sizeof(double)));

    // std::cout << "calling the kernel with " << num_blocks << " blocks" << std::endl;

    striped_warp_reduction_dot_kernel<<<num_blocks, num_threads>>>(
        num_rows, x_d, y_d, intermediate_sums_d
    );

    // CHECK_CUDA(hipDeviceSynchronize());

    // reduce_sums<<<1, num_threads>>>(intermediate_sums_d, num_blocks, result_d);

    CHECK_CUDA(hipDeviceSynchronize());

    // use thrust to reduce the intermediate sums
    // thrust::device_ptr<double> thrust_intermediate_sums(intermediate_sums_d);
    // double result = thrust::reduce(thrust_intermediate_sums, thrust_intermediate_sums + num_blocks, 0.0, thrust::plus<double>());


    // print intermediate sums
    // double * intermediate_sums = new double[num_blocks];
    // CHECK_CUDA(hipMemcpy(intermediate_sums, intermediate_sums_d, num_blocks * sizeof(double), hipMemcpyDeviceToHost));

    // for (int i = 0; i < num_blocks; i++){
    //     std::cout << "intermediate_sums[" << i << "] = " << intermediate_sums[i] << std::endl;
    // }

    // // write the result to the device
    // CHECK_CUDA(hipMemcpy(result_d, &result, sizeof(double), hipMemcpyHostToDevice));

    // use a kernel to reduce the intermediate sums
    reduce_sums<<<1, num_threads>>>(intermediate_sums_d, num_blocks, result_d);

    CHECK_CUDA(hipDeviceSynchronize());
    CHECK_CUDA(hipFree(intermediate_sums_d));

}

template class striped_warp_reduction_Implementation<double>;