#include "hip/hip_runtime.h"
#include "HPCG_versions/naiveBanded.cuh"
#include "UtilLib/utils.cuh"
#include <hip/hip_runtime.h>


__global__ void naiveBanded_SPMV_kernel(
        double* banded_A,
        int num_rows, int num_bands, int * j_min_i,
        double* x, double* y
    )
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // every thread computes one or more rows of the matrix
    for (int i = tid; i < num_rows; i += blockDim.x * gridDim.x) {
        // compute the matrix-vector product for the ith row
        double sum_i = 0;
        for (int band = 0; band < num_bands; band++) {
            int j = i + j_min_i[band];
            int current_row = i * num_bands;
            if (j >= 0 && j < num_rows) {
                sum_i += banded_A[current_row + band] * x[j];
            }
        }
        y[i] = sum_i;
    }
}
    

template <typename T>
void naiveBanded_Implementation<T>::naiveBanded_computeSPMV(
        banded_Matrix<T>& A, //we only pass A for the metadata
        T * banded_A_d, // the data of matrix A is already on the device
        int num_rows, int num_cols, // these refer to the shape of the banded matrix
        int num_bands, // the number of bands in the banded matrix
        int * j_min_i, // this is a mapping for calculating the j of some entry i,j in the banded matrix
        T * x_d, T * y_d // the vectors x and y are already on the device
    ) {
        // call the kernel for the naive banded SPMV
        // since every thread is working on one or more rows we need to base the number of threads on that
        int num_threads = NUM_CORES_PER_SM * 4;
        int num_blocks = std::min(NUM_PHYSICAL_CORES, ceiling_division(num_rows, num_threads));

        assert(num_bands == A.get_num_bands());
        assert(num_rows == A.get_num_rows());
        assert(num_cols == A.get_num_cols());

        // call the kernel
        naiveBanded_SPMV_kernel<<<num_blocks, num_threads>>>(
            banded_A_d, num_rows, num_bands, j_min_i, x_d, y_d
        );

        // synchronize the device
        hipDeviceSynchronize();
    }

// explicit template instantiation
template class naiveBanded_Implementation<double>;