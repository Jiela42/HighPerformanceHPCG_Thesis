#include "hip/hip_runtime.h"
#include "HPCG_versions/striped_COR_box_coloring.cuh"
#include "HPCG_versions/striped_coloring.cuh"
#include "UtilLib/cuda_utils.hpp"
#include "UtilLib/utils.cuh"
#include "MatrixLib/coloring.cuh"


// note: this function uses the same kernel as the implementation striped_coloring (hence the import)
// the difference is that this version does not need to compute the coloring, it is already precomputed
template <typename T>
void striped_COR_box_coloring_Implementation<T>::striped_COR_box_coloring_computeSymGS(
    striped_Matrix<T> & A,
    T * x_d, T * y_d // the vectors x and y are already on the device
){
    int diag_offset = A.get_diag_index();

    int num_rows = A.get_num_rows();
    int num_cols = A.get_num_cols();
    int num_stripes = A.get_num_stripes();
    int * j_min_i = A.get_j_min_i_d();
    T * striped_A_d = A.get_values_d();

    // the coloring was already computed, we can grab the pointers from the striped matrix object
    int * color_pointer_d = A.get_color_pointer_d();
    int * color_sorted_rows_d = A.get_color_sorted_rows_d();

    assert(diag_offset >= 0);
    // this assertion is here such that we don't benchmark the coloring computation
    // usually if these pointers are null we can just call the generate coloring function on the matrix
    assert(color_pointer_d != nullptr);
    assert(color_sorted_rows_d != nullptr);
    
    int nx = A.get_nx();
    int ny = A.get_ny();
    int nz = A.get_nz();

    // figure out how many colors we have at most (color zero has the most rows)
    int bx = this->bx;
    int by = this->by;
    int bz = this->bz;

    int num_color_cols = nx / bx;
    int num_color_rows = ny / by;
    int num_color_faces = nz / bz;

    num_color_cols = (0 < nx % bx) ? (num_color_cols + 1) : num_color_cols;
    num_color_rows = (0 < ny % by) ? (num_color_rows + 1) : num_color_rows;
    num_color_faces = (0 < nz % bz) ? (num_color_faces + 1) : num_color_faces;

    int max_num_rows_per_color = num_color_cols * num_color_rows * num_color_faces;
    int max_color = 26;

    // std::cout << "num_rows = " << num_rows << std::endl;
    // std::cout << "nx = " << nx << std::endl;
    // std::cout << "ny = " << ny << std::endl;
    // std::cout << "nz = " << nz << std::endl;


    int num_blocks = std::min(ceiling_division(max_num_rows_per_color, 1024/WARP_SIZE), MAX_NUM_BLOCKS);
    
    int max_iterations = this->max_SymGS_iterations;
    // std::cout << "max_iterations = " << max_iterations << std::endl;
    double norm0 = 1.0;
    double normi = norm0;

    if(max_iterations != 1){
        // compute the original L2 norm
        norm0 = this->L2_norm_for_SymGS(A, x_d, y_d);
    }
    
    for(int i = 0; i < max_iterations && normi/norm0 > this->SymGS_tolerance; i++){


        for(int color = 0; color <= max_color; color++){
            // we need to do a forward pass
            striped_coloring_half_SymGS_kernel<<<num_blocks, 1024>>>(
            color, color_pointer_d, color_sorted_rows_d,
            num_rows, num_cols,
            num_stripes, diag_offset,
            j_min_i,
            striped_A_d,
            x_d, y_d
            );
            CHECK_CUDA(hipDeviceSynchronize());
        }
    
        // we need to do a backward pass,
        // the colors for this are the same just in reverse order
        
        for(int color = max_color; color  >= 0; color--){
    
            striped_coloring_half_SymGS_kernel<<<num_blocks, 1024>>>(
            color, color_pointer_d, color_sorted_rows_d,
            num_rows, num_cols,
            num_stripes, diag_offset,
            j_min_i,
            striped_A_d,
            x_d, y_d
            );
            CHECK_CUDA(hipDeviceSynchronize());
        }

        if(max_iterations != 1){
            normi = this->L2_norm_for_SymGS(A, x_d, y_d);
        }
    }

}

// explicit template instantiation
template class striped_COR_box_coloring_Implementation<double>;